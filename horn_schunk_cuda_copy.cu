#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include <filesystem>
#include <tuple>

#ifdef USE_HIP

#include <hip/hip_runtime.h>
#include <iostream>
#define hipGetDeviceCount     hipGetDeviceCount
#define hipSetDevice          hipSetDevice
#define hipDeviceSynchronize  hipDeviceSynchronize

#define hipMalloc              hipMalloc 
#define hipFree                hipFree

#define cudaHostMalloc           hipHostMalloc
#define hipMemcpy              hipMemcpy

#define hipMemcpyHostToDevice  hipMemcpyHostToDevice
#define hipMemcpyDeviceToHost  hipMemcpyDeviceToHost

#define hipError_t             hipError_t
#define hipStream_t            hipStream_t
#define hipStreamCreate        hipStreamCreate
#define hipStreamDestroy       hipStreamDestroy
#define hipStreamSynchronize   hipStreamSynchronize
#define hipMemcpyAsync         hipMemcpyAsync
#define hipMemset              hipMemset
#define hipHostFree            hipHostFree
#define hipSuccess             hipSuccess
#define hipEventCreate         hipEventCreate
#define hipEventRecord         hipEventRecord
#define hipEventSynchronize    hipEventSynchronize
#define hipEventElapsedTime    hipEventElapsedTime
#define hipEventDestroy        hipEventDestroy
#define hipEvent_t             hipEvent_t

#else

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#endif

using namespace cv;
using namespace std;

// Visualize optical flow
void drawOpticalFlow(const Mat& flowX, const Mat& flowY, Mat& image, int scale = 3, int step = 16) {
    for (int y = 0; y < image.rows; y += step) {
        for (int x = 0; x < image.cols; x += step) {
            Point2f flow(flowX.at<double>(y, x), flowY.at<double>(y, x));
            Point start(x, y);
            Point end(cvRound(x + flow.x * scale), cvRound(y + flow.y * scale));
            arrowedLine(image, start, end, Scalar(0, 255, 0), 1, LINE_AA, 0, 0.2);
        }
    }
}

// Add this new visualization function
void visualizeFlowHSV(const Mat& flowU, const Mat& flowV, Mat& output) {
    Mat magnitude, angle;
    Mat hsv(flowU.size(), CV_8UC3);

    // Calculate magnitude and angle
    cartToPolar(flowU, flowV, magnitude, angle, true);

    // Normalize magnitude to the range [0, 255]
    normalize(magnitude, magnitude, 0, 255, NORM_MINMAX);

    // Create separate channels
    vector<Mat> channels(3);

    // H = angle (hue represents direction)
    angle.convertTo(channels[0], CV_8U, 180.0 / CV_PI / 2.0);  // Scale to [0, 180] for OpenCV

    // S = 255 (full saturation)
    channels[1] = Mat::ones(flowU.size(), CV_8U) * 255;

    // V = normalized magnitude
    magnitude.convertTo(channels[2], CV_8U);

    // Merge channels
    merge(channels, hsv);

    // Convert HSV to BGR
    cvtColor(hsv, output, COLOR_HSV2BGR);
}

template <typename T>
vector<T> matToVector(const Mat& mat) {
    if (mat.empty()) {
        throw runtime_error("Input matrix is empty.");
    }

    vector<T> vec(mat.rows * mat.cols * mat.channels());
    for (int y = 0; y < mat.rows; ++y) {
        const T* rowPtr = mat.ptr<T>(y);
        copy(rowPtr, rowPtr + mat.cols, vec.begin() + y * mat.cols);
    }
    return vec;
}

// Function to convert a std::vector back to cv::Mat
template <typename T>
Mat vectorToMat(const vector<T>& vec, int rows, int cols, int type) {
    Mat mat(rows, cols, type);
    for (int y = 0; y < rows; ++y) {
        T* rowPtr = mat.ptr<T>(y);
        copy(vec.begin() + y * cols, vec.begin() + (y + 1) * cols, rowPtr);
    }
    return mat;
}

void computeDerivatives(const Mat& im1, const Mat& im2, Mat& ix, Mat& iy, Mat& it) {
    // Define kernels for calculating derivatives
    Mat kernelX = (Mat_<double>(2, 2) << 0.25, -0.25, 0.25, -0.25); // Kernel for dx
    Mat kernelY = (Mat_<double>(2, 2) << 0.25, 0.25, -.25, -.25); // Kernel for dy
    Mat kernelT = (Mat_<double>(2, 2) << 0.25, 0.25, 0.25, 0.25);   // Kernel for dt

    // Convert images to double precision
    Mat im1_d, im2_d;
    im1.convertTo(im1_d, CV_64FC1);
    im2.convertTo(im2_d, CV_64FC1);

    // Compute derivatives
    Mat fx1, fx2, fy1, fy2, ft1, ft2;
    filter2D(im1_d, fx1, -1, kernelX);
    filter2D(im2_d, fx2, -1, kernelX);
    ix = fx1 + fx2;

    filter2D(im1_d, fy1, -1, kernelY);
    filter2D(im2_d, fy2, -1, kernelY);
    iy = fy1 + fy2;

    filter2D(im2_d, ft1, -1, -kernelT);
    filter2D(im1_d, ft2, -1, kernelT);
    it = ft1 + ft2;
}

__global__ void fused_horn_schunk(double* __restrict__ u, double* __restrict__ v,
                                 double* __restrict__ Ix, double* __restrict__ Iy, 
                                 double* __restrict__ It,
                                 double alpha, const int nx, const int ny) {     
    // Define halo width
    constexpr int HALO = 1;

    // Shared memory dimensions including halos
    __shared__ double s_u[18][18];
    __shared__ double s_v[18][18];
    
    // Global indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Local indices within the shared memory block
    int tx = threadIdx.x + HALO;
    int ty = threadIdx.y + HALO;
    
    // Global linear index
    int global_idx = y * nx + x;
    
    // Load center data
    if (x < nx && y < ny) {
        s_u[ty][tx] = u[global_idx];
        s_v[ty][tx] = v[global_idx];
    }
    
    // Load halo data
    // Top halo
    if (threadIdx.y == 0 && y > 0) {
        s_u[ty-HALO][tx] = u[global_idx - nx];
        s_v[ty-HALO][tx] = v[global_idx - nx];
    }
    
    // Bottom halo
    if (threadIdx.y == blockDim.y - 1 && y < ny - 1) {
        s_u[ty+HALO][tx] = u[global_idx + nx];
        s_v[ty+HALO][tx] = v[global_idx + nx];
    }
    
    // Left halo
    if (threadIdx.x == 0 && x > 0) {
        s_u[ty][tx-HALO] = u[global_idx - 1];
        s_v[ty][tx-HALO] = v[global_idx - 1];
    }
    
    // Right halo
    if (threadIdx.x == blockDim.x - 1 && x < nx - 1) {
        s_u[ty][tx+HALO] = u[global_idx + 1];
        s_v[ty][tx+HALO] = v[global_idx + 1];
    }
    
    // Corner halos
    if (threadIdx.x == 0 && threadIdx.y == 0 && x > 0 && y > 0) {
        s_u[ty-HALO][tx-HALO] = u[global_idx - nx - 1];
        s_v[ty-HALO][tx-HALO] = v[global_idx - nx - 1];
    }
    
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && x < nx - 1 && y > 0) {
        s_u[ty-HALO][tx+HALO] = u[global_idx - nx + 1];
        s_v[ty-HALO][tx+HALO] = v[global_idx - nx + 1];
    }
    
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && x > 0 && y < ny - 1) {
        s_u[ty+HALO][tx-HALO] = u[global_idx + nx - 1];
        s_v[ty+HALO][tx-HALO] = v[global_idx + nx - 1];
    }
    
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && x < nx - 1 && y < ny - 1) {
        s_u[ty+HALO][tx+HALO] = u[global_idx + nx + 1];
        s_v[ty+HALO][tx+HALO] = v[global_idx + nx + 1];
    }
    
    // Synchronize to ensure all data is loaded
    __syncthreads();
    
    // Compute flow updates only for interior points
    if (x > 0 && x < nx - 1 && y > 0 && y < ny - 1) {
        // First compute weighted averages
        double uAvg = (
            s_u[ty-1][tx-1] / 12.0 + 
            s_u[ty-1][tx]   / 6.0  + 
            s_u[ty-1][tx+1] / 12.0 + 
            s_u[ty][tx-1]   / 6.0  + 
            s_u[ty][tx+1]   / 6.0  + 
            s_u[ty+1][tx-1] / 12.0 + 
            s_u[ty+1][tx]   / 6.0  + 
            s_u[ty+1][tx+1] / 12.0
        );
        
        double vAvg = (
            s_v[ty-1][tx-1] / 12.0 + 
            s_v[ty-1][tx]   / 6.0  + 
            s_v[ty-1][tx+1] / 12.0 + 
            s_v[ty][tx-1]   / 6.0  + 
            s_v[ty][tx+1]   / 6.0  + 
            s_v[ty+1][tx-1] / 12.0 + 
            s_v[ty+1][tx]   / 6.0  + 
            s_v[ty+1][tx+1] / 12.0
        );

        // Then compute Horn-Schunck update
        double ix = Ix[global_idx];
        double iy = Iy[global_idx];
        double it = It[global_idx];

        double denom = alpha * alpha + ix * ix + iy * iy;
        double p = (ix * uAvg + iy * vAvg + it);
        
        // Write final results directly
        u[global_idx] = uAvg - ix * (p / denom);
        v[global_idx] = vAvg - iy * (p / denom);
    }
}

// Hardware specifications for roofline model
const double PEAK_MEMORY_BANDWIDTH = 400e9;  // 400 GB/s
const double PEAK_FLOP_RATE = 2.5e12;       // 2.5 TFLOP/s

// Calculate theoretical peak performance based on arithmetic intensity
double calculate_roofline(double arithmetic_intensity) {
    return min(PEAK_FLOP_RATE, PEAK_MEMORY_BANDWIDTH * arithmetic_intensity);
}

// Modify the analyze_performance function to return the metrics
tuple<double, double, double> analyze_performance(int nx, int ny, int iterations, double elapsed_time) {
    size_t total_pixels = nx * ny;
    
    // Fused kernel FLOPs per pixel:
    // - Averages computation: 22 FLOPs
    // - Horn-Schunck update: 13 FLOPs
    // Total: 35 FLOPs per pixel
    // NOT CORRECT
    double flops_per_iteration = 35.0 * total_pixels;
    double total_flops = flops_per_iteration * iterations;
    
    // Memory operations per iteration:
    // - Reads: u, v, Ix, Iy, It (5 doubles)
    // - Writes: u_new, v_new (2 doubles)
    // Total: 7 doubles per pixel per iteration
    // NOT CORRECT
    size_t bytes_per_pixel_per_iteration = 7 * sizeof(double);
    double total_bytes = total_pixels * bytes_per_pixel_per_iteration * iterations;
    
    // Calculate metrics
    double arithmetic_intensity = total_flops / total_bytes;
    double achieved_tflops = total_flops / (elapsed_time * 1e12);
    double peak_tflops = calculate_roofline(arithmetic_intensity) / 1e12;
    
    return make_tuple(achieved_tflops, arithmetic_intensity, peak_tflops);
}

// Main function demonstrating usage
int main(int argc, char* argv[]) {
    cout << "Running Horn-Schunck optical flow..." << endl;

    string filename1 = argv[1];
    string filename2 = argv[2];
    string outputname = argv[3];

    // Load two consecutive frames
    Mat frame1_original = imread(filename1, 0);
    Mat frame2_original = imread(filename2, 0);
   
    if (frame1_original.empty() || frame2_original.empty()) {
        cerr << "Error loading images!" << endl;
        cerr << "Make sure " << filename1 << " and " << filename2 << " exist in: " << filesystem::current_path() << endl;
        return -1;
    }

    // Define sizes to test
    vector<int> test_sizes = {128, 256, 512};  // Test smaller sizes first
    
    for (int size : test_sizes) {
        cout << "\nProcessing size: " << size << "x" << size << endl;
        
        // Resize images
        Mat frame1, frame2;
        resize(frame1_original, frame1, Size(size, size));
        resize(frame2_original, frame2, Size(size, size));
        
        cout << "Resized images - Frame1: " << frame1.size() << " Frame2: " << frame2.size() << endl;
        
        // Image size and grid sizes
        hipError_t GPU_ERROR;
        int ny = frame1.rows;
        int nx = frame1.cols;
        size_t size_bytes = nx * ny * sizeof(double);

        // Compute image derivatives
        Mat IxMat, IyMat, ItMat;
        computeDerivatives(frame1, frame2, IxMat, IyMat, ItMat);

        // Convert derivatives to vectors
        vector<double> IxHost = matToVector<double>(IxMat);
        vector<double> IyHost = matToVector<double>(IyMat);
        vector<double> ItHost = matToVector<double>(ItMat);

        double *IxDevice, *IyDevice, *ItDevice;
        // Allocate memory on device
        GPU_ERROR = hipMalloc(&IxDevice, size_bytes);
        if (GPU_ERROR != hipSuccess) {
            cerr << "Failed to allocate Ix device memory" << endl;
            continue;  // Skip to next size if allocation fails
        }
        GPU_ERROR = hipMalloc(&IyDevice, size_bytes);
        if (GPU_ERROR != hipSuccess) {
            hipFree(IxDevice);
            cerr << "Failed to allocate Iy device memory" << endl;
            continue;
        }
        GPU_ERROR = hipMalloc(&ItDevice, size_bytes);
        if (GPU_ERROR != hipSuccess) {
            hipFree(IxDevice);
            hipFree(IyDevice);
            cerr << "Failed to allocate It device memory" << endl;
            continue;
        }

        // Copy derivatives to device
        GPU_ERROR = hipMemcpy(IxDevice, IxHost.data(), size_bytes, hipMemcpyHostToDevice);
        GPU_ERROR = hipMemcpy(IyDevice, IyHost.data(), size_bytes, hipMemcpyHostToDevice);
        GPU_ERROR = hipMemcpy(ItDevice, ItHost.data(), size_bytes, hipMemcpyHostToDevice);

        // Allocate device memory for results
        double *u_d, *v_d;
        GPU_ERROR = hipMalloc(&u_d, size_bytes);
        GPU_ERROR = hipMalloc(&v_d, size_bytes);

        // Initialize device memory to zero
        GPU_ERROR = hipMemset(u_d, 0, size_bytes);
        GPU_ERROR = hipMemset(v_d, 0, size_bytes);

        // Set up grid and block dimensions
        int BLOCK_DIM_X = 16;
        int BLOCK_DIM_Y = 16;
        dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
        dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

        // Add timing variables
        hipEvent_t start, stop;
        GPU_ERROR = hipEventCreate(&start);
        GPU_ERROR = hipEventCreate(&stop);
        
        // Start timing
        GPU_ERROR = hipEventRecord(start);

        // Compute optical flow with fused kernel
        int iterations = 200;
        double alpha = 1;
        
        for(int iter = 0; iter < iterations; iter++) {
            fused_horn_schunk<<<grid, block>>>(u_d, v_d, IxDevice, IyDevice, ItDevice, alpha, nx, ny);
        }
        
        // Stop timing
        GPU_ERROR = hipEventRecord(stop);
        GPU_ERROR = hipEventSynchronize(stop);
        
        float elapsed_time_ms;
        GPU_ERROR = hipEventElapsedTime(&elapsed_time_ms, start, stop);
        double elapsed_time_s = elapsed_time_ms / 1000.0;
        
        // Get performance metrics
        auto [measured_tflops, ai, peak_tflops] = analyze_performance(nx, ny, iterations, elapsed_time_s);
        
        // Print performance table
        printf("\nGrid Size | TFLOPS | AI | Peak TFLOPS | Time(s) | Iterations\n");
        printf("---------|--------|----|-----------  |---------|------------|\n");
        printf("%4dx%4d | %6.6f | %6.6f | %6.6f | %7.6f | %10d |\n",
               nx, ny, measured_tflops, ai, peak_tflops, elapsed_time_s, iterations);

        // Copy results back to host
        vector<double> u_result(nx * ny);
        vector<double> v_result(nx * ny);
        GPU_ERROR = hipMemcpy(u_result.data(), u_d, size_bytes, hipMemcpyDeviceToHost);
        GPU_ERROR = hipMemcpy(v_result.data(), v_d, size_bytes, hipMemcpyDeviceToHost);

        // Visualize and save results
        Mat img_color, flowX, flowY;
        cvtColor(frame1, img_color, COLOR_GRAY2BGR);
        flowX = vectorToMat<double>(u_result, ny, nx, CV_64F);
        flowY = vectorToMat<double>(v_result, ny, nx, CV_64F); 
        drawOpticalFlow(flowX, flowY, img_color);

        Mat flow_vis;
        visualizeFlowHSV(flowX, flowY, flow_vis);

        string size_str = to_string(size);
        imwrite("outputs/CUDA_optical_flow_" + outputname + "_" + size_str + "x" + size_str + ".png", img_color);
        imwrite("outputs/CUDA_optical_flow_hsv_" + outputname + "_" + size_str + "x" + size_str + ".png", flow_vis);

        // Cleanup
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(IxDevice);
        hipFree(IyDevice);
        hipFree(ItDevice);
        hipFree(u_d);
        hipFree(v_d);
        
        // Force synchronization before next iteration
        hipDeviceSynchronize();
    }

    return 0;
}