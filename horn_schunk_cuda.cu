#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include <filesystem>

#ifdef USE_HIP

#include <hip/hip_runtime.h>
#include <iostream>
#define hipGetDeviceCount     hipGetDeviceCount
#define hipSetDevice          hipSetDevice
#define hipDeviceSynchronize  hipDeviceSynchronize


#define hipMalloc              hipMalloc 
#define hipFree                hipFree

#define cudaHostMalloc           hipHostMalloc
#define hipMemcpy              hipMemcpy

#define hipMemcpyHostToDevice  hipMemcpyHostToDevice
#define hipMemcpyDeviceToHost  hipMemcpyDeviceToHost

#define hipError_t             hipError_t

#else

#include <hip/hip_runtime.h>

#endif

using namespace cv;
using namespace std;


// Visualize optical flow
void drawOpticalFlow(const Mat& flowX, const Mat& flowY, Mat& image, int scale = 3, int step = 16) {
    for (int y = 0; y < image.rows; y += step) {
        for (int x = 0; x < image.cols; x += step) {
            Point2f flow(flowX.at<double>(y, x), flowY.at<double>(y, x));
            Point start(x, y);
            Point end(cvRound(x + flow.x * scale), cvRound(y + flow.y * scale));
            arrowedLine(image, start, end, Scalar(0, 255, 0), 1, LINE_AA, 0, 0.2);
        }
    }
}

// Add this new visualization function
void visualizeFlowHSV(const Mat& flowU, const Mat& flowV, Mat& output) {
    Mat magnitude, angle;
    Mat hsv(flowU.size(), CV_8UC3);

    // Calculate magnitude and angle
    cartToPolar(flowU, flowV, magnitude, angle, true);

    // Normalize magnitude to the range [0, 255]
    normalize(magnitude, magnitude, 0, 255, NORM_MINMAX);

    // Create separate channels
    vector<Mat> channels(3);

    // H = angle (hue represents direction)
    angle.convertTo(channels[0], CV_8U, 180.0 / CV_PI / 2.0);  // Scale to [0, 180] for OpenCV

    // S = 255 (full saturation)
    channels[1] = Mat::ones(flowU.size(), CV_8U) * 255;

    // V = normalized magnitude
    magnitude.convertTo(channels[2], CV_8U);

    // Merge channels
    merge(channels, hsv);

    // Convert HSV to BGR
    cvtColor(hsv, output, COLOR_HSV2BGR);
}

template <typename T>
vector<T> matToVector(const Mat& mat) {
    if (mat.empty()) {
        throw runtime_error("Input matrix is empty.");
    }

    vector<T> vec(mat.rows * mat.cols * mat.channels());
    for (int y = 0; y < mat.rows; ++y) {
        const T* rowPtr = mat.ptr<T>(y);
        copy(rowPtr, rowPtr + mat.cols, vec.begin() + y * mat.cols);
    }
    return vec;
}

// Function to convert a std::vector back to cv::Mat
template <typename T>
Mat vectorToMat(const vector<T>& vec, int rows, int cols, int type) {
    Mat mat(rows, cols, type);
    for (int y = 0; y < rows; ++y) {
        T* rowPtr = mat.ptr<T>(y);
        copy(vec.begin() + y * cols, vec.begin() + (y + 1) * cols, rowPtr);
    }
    return mat;
}

void computeDerivatives(const Mat& im1, const Mat& im2, Mat& ix, Mat& iy, Mat& it) {
    // Define kernels for calculating derivatives
    Mat kernelX = (Mat_<double>(2, 2) << 0.25, -0.25, 0.25, -0.25); // Kernel for dx
    Mat kernelY = (Mat_<double>(2, 2) << 0.25, 0.25, -.25, -.25); // Kernel for dy
    Mat kernelT = (Mat_<double>(2, 2) << 0.25, 0.25, 0.25, 0.25);   // Kernel for dt

    // Convert images to double precision
    Mat im1_d, im2_d;
    im1.convertTo(im1_d, CV_64FC1);
    im2.convertTo(im2_d, CV_64FC1);

    // Compute derivatives
    Mat fx1, fx2, fy1, fy2, ft1, ft2;
    filter2D(im1_d, fx1, -1, kernelX);
    filter2D(im2_d, fx2, -1, kernelX);
    ix = fx1 + fx2;

    filter2D(im1_d, fy1, -1, kernelY);
    filter2D(im2_d, fy2, -1, kernelY);
    iy = fy1 + fy2;

    filter2D(im2_d, ft1, -1, -kernelT);
    filter2D(im1_d, ft2, -1, kernelT);
    it = ft1 + ft2;
}

__global__ void compute_neighbor_average(double* __restrict__ u, double* __restrict__ v, 
                            double* __restrict__ uAvg, double* __restrict__ vAvg,
                               const int nx, const int ny) {     
    __shared__ double shared_u[18][18];
    __shared__ double shared_v[18][18];

   // Thread indices
    const int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Local thread indices
    const int local_x = threadIdx.x;
    const int local_y = threadIdx.y;
    
    // Shared memory dimensions including halos
    const int shared_idx = local_y + 1;  // +1 to avoid the halo index at 0
    const int shared_jdx = local_x + 1;  // +1 to avoid the halo index at 0

    // Global index for linear arrays
    const int global_idx = global_y * nx + global_x;

    // Load main region into shared memory (including boundary elements)
    if (global_x < nx && global_y < ny) {
        shared_u[shared_jdx][shared_idx] = u[global_idx];
        shared_v[shared_jdx][shared_idx] = v[global_idx];
    } else {
        shared_u[shared_jdx][shared_idx] = 0.0;
        shared_v[shared_jdx][shared_idx] = 0.0;
    }

    // Load halos for neighboring elements from global memory
    if (local_x == 0 && global_x > 0) {
        shared_u[shared_jdx - 1][shared_idx] = u[global_idx - 1];
        shared_v[shared_jdx - 1][shared_idx] = v[global_idx - 1];
    }
    if (local_x == blockDim.x - 1 && global_x < nx - 1) {
        shared_u[shared_jdx + 1][shared_idx] = u[global_idx + 1];
        shared_v[shared_jdx + 1][shared_idx] = v[global_idx + 1];
    }
    if (local_y == 0 && global_y > 0) {
        shared_u[shared_jdx][shared_idx - 1] = u[global_idx - nx];
        shared_v[shared_jdx][shared_idx - 1] = v[global_idx - nx];
    }
    if (local_y == blockDim.y - 1 && global_y < ny - 1) {
        shared_u[shared_jdx][shared_idx + 1] = u[global_idx + nx];
        shared_v[shared_jdx][shared_idx + 1] = v[global_idx + nx];
    }

    // Corners (Halo corners)
    if (local_x == 0 && local_y == 0 && global_x > 0 && global_y > 0) {
        shared_u[shared_jdx - 1][shared_idx - 1] = u[global_idx - nx - 1];
        shared_v[shared_jdx - 1][shared_idx - 1] = v[global_idx - nx - 1];
    }
    if (local_x == blockDim.x - 1 && local_y == 0 && global_x < nx - 1 && global_y > 0) {
        shared_u[shared_jdx + 1][shared_idx - 1] = u[global_idx - nx + 1];
        shared_v[shared_jdx + 1][shared_idx - 1] = v[global_idx - nx + 1];
    }
    if (local_x == 0 && local_y == blockDim.y - 1 && global_x > 0 && global_y < ny - 1) {
        shared_u[shared_jdx - 1][shared_idx + 1] = u[global_idx + nx - 1];
        shared_v[shared_jdx - 1][shared_idx + 1] = v[global_idx + nx - 1];
    }
    if (local_x == blockDim.x - 1 && local_y == blockDim.y - 1 && global_x < nx - 1 && global_y < ny - 1) {
        shared_u[shared_jdx + 1][shared_idx + 1] = u[global_idx + nx + 1];
        shared_v[shared_jdx + 1][shared_idx + 1] = v[global_idx + nx + 1];
    }

    // Synchronize to ensure all threads have loaded shared memory
    __syncthreads();

    // Compute neighbor average for non-border threads
    if (global_x > 0 && global_x < nx - 1 && global_y > 0 && global_y < ny - 1) {
        uAvg[global_idx] = (
            shared_u[shared_jdx - 1][shared_idx - 1] / 12 + shared_u[shared_jdx][shared_idx - 1] / 6 + shared_u[shared_jdx + 1][shared_idx - 1] / 12 +
            shared_u[shared_jdx - 1][shared_idx] / 6 + shared_u[shared_jdx + 1][shared_idx] / 6 +
            shared_u[shared_jdx - 1][shared_idx + 1] / 12 + shared_u[shared_jdx][shared_idx + 1] / 6 + shared_u[shared_jdx + 1][shared_idx + 1] / 12
        );

        vAvg[global_idx] = (
            shared_v[shared_jdx - 1][shared_idx - 1] / 12 + shared_v[shared_jdx][shared_idx - 1] / 6 + shared_v[shared_jdx + 1][shared_idx - 1] / 12 +
            shared_v[shared_jdx - 1][shared_idx] / 6 + shared_v[shared_jdx + 1][shared_idx] / 6 +
            shared_v[shared_jdx - 1][shared_idx + 1] / 12 + shared_v[shared_jdx][shared_idx + 1] / 6 + shared_v[shared_jdx + 1][shared_idx + 1] / 12
        );
    }
}

__global__ void horn_schunk(double* __restrict__ u, double* __restrict__ v, 
                            double* __restrict__ uAvg, double* __restrict__ vAvg,
                            double* __restrict__ Ix, double* __restrict__ Iy, double* __restrict__ It,
                               double alpha, const int nx, const int ny) { 
    const int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int global_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int idx = global_y * nx + global_x;  
    
    if (global_x < nx && global_y < ny) {
        double ix = Ix[idx];
        double iy = Iy[idx];
        double it = It[idx];
        double uAvgVal = uAvg[idx];
        double vAvgVal = vAvg[idx];

        double denom = alpha * alpha + ix * ix + iy * iy;
        double p = (ix * uAvgVal + iy * vAvgVal + it);
        u[idx] = uAvgVal - ix * (p / denom);
        v[idx] = vAvgVal - iy * (p / denom);
    }
}

// Main function demonstrating usage
int main(int argc, char* argv[]) {
    cout << "Running Horn-Schunck optical flow..." << endl;

    string filename1 = argv[1];
    string filename2 = argv[2];
    string outputname = argv[3];

    // Load two consecutive frames
    Mat frame1 = imread(filename1, 0);
    Mat frame2 = imread(filename2, 0);
   
    if (frame1.empty() || frame2.empty()) {
        cerr << "Error loading images!" << endl;
        cerr << "Make sure " << filename1 << " and " << filename2 << " exist in: " << filesystem::current_path() << endl;
        return -1;
    }
   
    cout << "Loaded images - Frame1: " << frame1.size() << " Frame2: " << frame2.size() << endl;
    
    // Image size and grid sizes
    hipError_t GPU_ERROR;
    int ny = frame1.rows;
    int nx = frame1.cols;
    size_t size = nx * ny * sizeof(double);
    int BLOCK_DIM_X = 16;
    int BLOCK_DIM_Y = 16;
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid((nx + block.x - 1) / block.x,
              (ny + block.y - 1) / block.y);
    cout << "grid x dim:" << (nx + block.x - 1) / block.x << ", grid y dim:" << (ny + block.y - 1) / block.y << endl;

   // Compute image derivatives
    Mat IxMat, IyMat, ItMat;
    computeDerivatives(frame1, frame2, IxMat, IyMat, ItMat);

    // Convert derivatives to vectors
    vector<double> IxHost = matToVector<double>(IxMat);
    vector<double> IyHost = matToVector<double>(IyMat);
    vector<double> ItHost = matToVector<double>(ItMat);

    // Copy derivatives to host
    double *IxDevice, *IyDevice, *ItDevice;
    GPU_ERROR = hipMalloc(&IxDevice, size);
    GPU_ERROR = hipMalloc(&IyDevice, size);
    GPU_ERROR = hipMalloc(&ItDevice, size);
    GPU_ERROR = hipMemcpy(IxDevice, IxHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(IyDevice, IyHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(ItDevice, ItHost.data(), size, hipMemcpyHostToDevice);
    cout << "Finished derivatives transfer" << endl;

    // Create average and flow vectors for device and host
    vector<double> uHost(nx * ny, 0.0);
    vector<double> vHost(nx * ny, 0.0);
    double *uDevice, *vDevice;
    GPU_ERROR = hipMalloc(&uDevice, size);
    GPU_ERROR = hipMalloc(&vDevice, size);
    GPU_ERROR = hipMemcpy(uDevice, uHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(vDevice, vHost.data(), size, hipMemcpyHostToDevice);

    double *uAverage, *vAverage;
    GPU_ERROR = hipMalloc(&uAverage, size);
    GPU_ERROR = hipMalloc(&vAverage, size);
    GPU_ERROR = hipMemcpy(uAverage, uHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(vAverage, vHost.data(), size, hipMemcpyHostToDevice);
    cout << "Copied over average and flow vectors" << endl;

    // Compute optical flow
    int currIteration = 0;
    int iterations = 200;
    double alpha = 1;
    while (currIteration < iterations){
        compute_neighbor_average<<<grid, block>>>(uDevice, vDevice, uAverage, vAverage, nx, ny);      
        GPU_ERROR = hipDeviceSynchronize();

        horn_schunk<<<grid, block>>>(uDevice, vDevice, uAverage, vAverage, IxDevice, IyDevice, ItDevice, alpha, nx, ny);
        GPU_ERROR = hipDeviceSynchronize();

        currIteration++;
    }
    cout << "Kernels finished" << endl;

    // Copy over flow results to host
    GPU_ERROR = hipMemcpy(uHost.data(), uDevice, size, hipMemcpyDeviceToHost);
    GPU_ERROR = hipMemcpy(vHost.data(), vDevice, size, hipMemcpyDeviceToHost);
    cout << "Copied results" << endl;

    // Visualize optical flow
    Mat img_color, flowX, flowY;
    cvtColor(frame1, img_color, COLOR_GRAY2BGR);
    flowX = vectorToMat<double>(uHost, ny, nx, CV_64F);
    flowY = vectorToMat<double>(vHost, ny, nx, CV_64F); 
    drawOpticalFlow(flowX, flowY, img_color);

    Mat flow_vis;
    visualizeFlowHSV(flowX, flowY, flow_vis);

    cout << "Writing optical flow images." << endl;
    imwrite("outputs/CUDA_optical_flow_" + outputname + ".png", img_color);
    imwrite("outputs/CUDA_optical_flow_hsv_" + outputname + ".png", flow_vis);
    return 0;
}