#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include <filesystem>

#ifdef USE_HIP

#include <hip/hip_runtime.h>
#include <iostream>
#define hipGetDeviceCount     hipGetDeviceCount
#define hipSetDevice          hipSetDevice
#define hipDeviceSynchronize  hipDeviceSynchronize


#define hipMalloc              hipMalloc 
#define hipFree                hipFree

#define cudaHostMalloc           hipHostMalloc
#define hipMemcpy              hipMemcpy

#define hipMemcpyHostToDevice  hipMemcpyHostToDevice
#define hipMemcpyDeviceToHost  hipMemcpyDeviceToHost

#define hipError_t             hipError_t

#else

#include <hip/hip_runtime.h>

#endif

using namespace cv;
using namespace std;


// Visualize optical flow
void drawOpticalFlow(const Mat& flowX, const Mat& flowY, Mat& image, int scale = 3, int step = 16) {
    for (int y = 0; y < image.rows; y += step) {
        for (int x = 0; x < image.cols; x += step) {
            Point2f flow(flowX.at<double>(y, x), flowY.at<double>(y, x));
            Point start(x, y);
            Point end(cvRound(x + flow.x * scale), cvRound(y + flow.y * scale));
            arrowedLine(image, start, end, Scalar(0, 255, 0), 1, LINE_AA, 0, 0.2);
        }
    }
}

// Add this new visualization function
void visualizeFlowHSV(const Mat& flowU, const Mat& flowV, Mat& output) {
    Mat magnitude, angle;
    Mat hsv(flowU.size(), CV_8UC3);

    // Calculate magnitude and angle
    cartToPolar(flowU, flowV, magnitude, angle, true);

    // Normalize magnitude to the range [0, 255]
    normalize(magnitude, magnitude, 0, 255, NORM_MINMAX);

    // Create separate channels
    vector<Mat> channels(3);

    // H = angle (hue represents direction)
    angle.convertTo(channels[0], CV_8U, 180.0 / CV_PI / 2.0);  // Scale to [0, 180] for OpenCV

    // S = 255 (full saturation)
    channels[1] = Mat::ones(flowU.size(), CV_8U) * 255;

    // V = normalized magnitude
    magnitude.convertTo(channels[2], CV_8U);

    // Merge channels
    merge(channels, hsv);

    // Convert HSV to BGR
    cvtColor(hsv, output, COLOR_HSV2BGR);
}

template <typename T>
vector<T> matToVector(const Mat& mat) {
    if (mat.empty()) {
        throw runtime_error("Input matrix is empty.");
    }

    vector<T> vec(mat.rows * mat.cols * mat.channels());
    for (int y = 0; y < mat.rows; ++y) {
        const T* rowPtr = mat.ptr<T>(y);
        copy(rowPtr, rowPtr + mat.cols, vec.begin() + y * mat.cols);
    }
    return vec;
}

// Function to convert a std::vector back to cv::Mat
template <typename T>
Mat vectorToMat(const vector<T>& vec, int rows, int cols, int type) {
    Mat mat(rows, cols, type);
    for (int y = 0; y < rows; ++y) {
        T* rowPtr = mat.ptr<T>(y);
        copy(vec.begin() + y * cols, vec.begin() + (y + 1) * cols, rowPtr);
    }
    return mat;
}

void computeDerivatives(const Mat& im1, const Mat& im2, Mat& ix, Mat& iy, Mat& it) {
    // Define kernels for calculating derivatives
    Mat kernelX = (Mat_<double>(2, 2) << 0.25, -0.25, 0.25, -0.25); // Kernel for dx
    Mat kernelY = (Mat_<double>(2, 2) << 0.25, 0.25, -.25, -.25); // Kernel for dy
    Mat kernelT = (Mat_<double>(2, 2) << 0.25, 0.25, 0.25, 0.25);   // Kernel for dt

    // Convert images to double precision
    Mat im1_d, im2_d;
    im1.convertTo(im1_d, CV_64FC1);
    im2.convertTo(im2_d, CV_64FC1);

    // Compute derivatives
    Mat fx1, fx2, fy1, fy2, ft1, ft2;
    filter2D(im1_d, fx1, -1, kernelX);
    filter2D(im2_d, fx2, -1, kernelX);
    ix = fx1 + fx2;

    filter2D(im1_d, fy1, -1, kernelY);
    filter2D(im2_d, fy2, -1, kernelY);
    iy = fy1 + fy2;

    filter2D(im2_d, ft1, -1, -kernelT);
    filter2D(im1_d, ft2, -1, kernelT);
    it = ft1 + ft2;
}

__global__ void compute_neighbor_average(double* __restrict__ u, double* __restrict__ v, 
                            double* __restrict__ uAvg, double* __restrict__ vAvg,
                               const int nx, const int ny) {     
    // Define halo width
    constexpr int HALO = 1;

    // Shared memory dimensions including halos
    __shared__ double s_u[18][18];
    __shared__ double s_v[18][18];
    
    // Global indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Local indices within the shared memory block
    int tx = threadIdx.x + HALO;
    int ty = threadIdx.y + HALO;
    
    // Global linear index
    int global_idx = y * nx + x;
    
    // Load center data
    if (x < nx && y < ny) {
        s_u[ty][tx] = u[global_idx];
        s_v[ty][tx] = v[global_idx];
    }
    
    // Load halo data
    // Top halo
    if (threadIdx.y == 0 && y > 0) {
        s_u[ty-HALO][tx] = u[global_idx - nx];
        s_v[ty-HALO][tx] = v[global_idx - nx];
    }
    
    // Bottom halo
    if (threadIdx.y == blockDim.y - 1 && y < ny - 1) {
        s_u[ty+HALO][tx] = u[global_idx + nx];
        s_v[ty+HALO][tx] = v[global_idx + nx];
    }
    
    // Left halo
    if (threadIdx.x == 0 && x > 0) {
        s_u[ty][tx-HALO] = u[global_idx - 1];
        s_v[ty][tx-HALO] = v[global_idx - 1];
    }
    
    // Right halo
    if (threadIdx.x == blockDim.x - 1 && x < nx - 1) {
        s_u[ty][tx+HALO] = u[global_idx + 1];
        s_v[ty][tx+HALO] = v[global_idx + 1];
    }
    
    // Corner halos
    // Top-left
    if (threadIdx.x == 0 && threadIdx.y == 0 && x > 0 && y > 0) {
        s_u[ty-HALO][tx-HALO] = u[global_idx - nx - 1];
        s_v[ty-HALO][tx-HALO] = v[global_idx - nx - 1];
    }
    
    // Top-right
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && x < nx - 1 && y > 0) {
        s_u[ty-HALO][tx+HALO] = u[global_idx - nx + 1];
        s_v[ty-HALO][tx+HALO] = v[global_idx - nx + 1];
    }
    
    // Bottom-left
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && x > 0 && y < ny - 1) {
        s_u[ty+HALO][tx-HALO] = u[global_idx + nx - 1];
        s_v[ty+HALO][tx-HALO] = v[global_idx + nx - 1];
    }
    
    // Bottom-right
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && x < nx - 1 && y < ny - 1) {
        s_u[ty+HALO][tx+HALO] = u[global_idx + nx + 1];
        s_v[ty+HALO][tx+HALO] = v[global_idx + nx + 1];
    }
    
    // Synchronize to ensure all data is loaded
    __syncthreads();
    
    // Compute averages only for interior threads
    if (x > 0 && x < nx - 1 && y > 0 && y < ny - 1) {
        // Compute uAvg using 3x3 neighborhood with weighted average
        uAvg[global_idx] = (
            s_u[ty-1][tx-1] / 12.0 + 
            s_u[ty-1][tx]   / 6.0  + 
            s_u[ty-1][tx+1] / 12.0 + 
            s_u[ty][tx-1]   / 6.0  + 
            s_u[ty][tx+1]   / 6.0  + 
            s_u[ty+1][tx-1] / 12.0 + 
            s_u[ty+1][tx]   / 6.0  + 
            s_u[ty+1][tx+1] / 12.0
        );
        
        // Compute vAvg using 3x3 neighborhood with weighted average
        vAvg[global_idx] = (
            s_v[ty-1][tx-1] / 12.0 + 
            s_v[ty-1][tx]   / 6.0  + 
            s_v[ty-1][tx+1] / 12.0 + 
            s_v[ty][tx-1]   / 6.0  + 
            s_v[ty][tx+1]   / 6.0  + 
            s_v[ty+1][tx-1] / 12.0 + 
            s_v[ty+1][tx]   / 6.0  + 
            s_v[ty+1][tx+1] / 12.0
        );
    }
}

__global__ void horn_schunk(double* __restrict__ u, double* __restrict__ v, 
                            double* __restrict__ uAvg, double* __restrict__ vAvg,
                            double* __restrict__ Ix, double* __restrict__ Iy, double* __restrict__ It,
                               double alpha, const int nx, const int ny) { 
    const int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int global_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int idx = global_y * nx + global_x;  
    
    if (global_x < nx && global_y < ny) {
        double ix = Ix[idx];
        double iy = Iy[idx];
        double it = It[idx];
        double uAvgVal = uAvg[idx];
        double vAvgVal = vAvg[idx];

        double denom = alpha * alpha + ix * ix + iy * iy;
        double p = (ix * uAvgVal + iy * vAvgVal + it);
        u[idx] = uAvgVal - ix * (p / denom);
        v[idx] = vAvgVal - iy * (p / denom);
    }
}

// Main function demonstrating usage
int main(int argc, char* argv[]) {
    cout << "Running Horn-Schunck optical flow..." << endl;

    string filename1 = argv[1];
    string filename2 = argv[2];
    string outputname = argv[3];

    // Load two consecutive frames
    Mat frame1 = imread(filename1, 0);
    Mat frame2 = imread(filename2, 0);
   
    if (frame1.empty() || frame2.empty()) {
        cerr << "Error loading images!" << endl;
        cerr << "Make sure " << filename1 << " and " << filename2 << " exist in: " << filesystem::current_path() << endl;
        return -1;
    }
   
    cout << "Loaded images - Frame1: " << frame1.size() << " Frame2: " << frame2.size() << endl;
    
    // Image size and grid sizes
    hipError_t GPU_ERROR;
    int ny = frame1.rows;
    int nx = frame1.cols;
    size_t size = nx * ny * sizeof(double);
    int BLOCK_DIM_X = 16;
    int BLOCK_DIM_Y = 16;
    dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 grid((nx + block.x - 1) / block.x,
              (ny + block.y - 1) / block.y);
    cout << "grid x dim:" << (nx + block.x - 1) / block.x << ", grid y dim:" << (ny + block.y - 1) / block.y << endl;

   // Compute image derivatives
    Mat IxMat, IyMat, ItMat;
    computeDerivatives(frame1, frame2, IxMat, IyMat, ItMat);

    // Convert derivatives to vectors
    vector<double> IxHost = matToVector<double>(IxMat);
    vector<double> IyHost = matToVector<double>(IyMat);
    vector<double> ItHost = matToVector<double>(ItMat);

    // Copy derivatives to host
    double *IxDevice, *IyDevice, *ItDevice;
    GPU_ERROR = hipMalloc(&IxDevice, size);
    GPU_ERROR = hipMalloc(&IyDevice, size);
    GPU_ERROR = hipMalloc(&ItDevice, size);
    GPU_ERROR = hipMemcpy(IxDevice, IxHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(IyDevice, IyHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(ItDevice, ItHost.data(), size, hipMemcpyHostToDevice);
    cout << "Finished derivatives transfer" << endl;

    // Create average and flow vectors for device and host
    vector<double> uHost(nx * ny, 0.0);
    vector<double> vHost(nx * ny, 0.0);
    double *uDevice, *vDevice;
    GPU_ERROR = hipMalloc(&uDevice, size);
    GPU_ERROR = hipMalloc(&vDevice, size);
    GPU_ERROR = hipMemcpy(uDevice, uHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(vDevice, vHost.data(), size, hipMemcpyHostToDevice);

    double *uAverage, *vAverage;
    GPU_ERROR = hipMalloc(&uAverage, size);
    GPU_ERROR = hipMalloc(&vAverage, size);
    GPU_ERROR = hipMemcpy(uAverage, uHost.data(), size, hipMemcpyHostToDevice);
    GPU_ERROR = hipMemcpy(vAverage, vHost.data(), size, hipMemcpyHostToDevice);
    cout << "Copied over average and flow vectors" << endl;

    // Add near the top of main, after device allocation setup
    const int NUM_STREAMS = 2;  // You can adjust this number
    hipStream_t streams[NUM_STREAMS];
    
    // Calculate sizes for each stream
    int rows_per_stream = ny / NUM_STREAMS;
    size_t stream_size = nx * rows_per_stream * sizeof(double);
    
    // Arrays of device pointers for each stream
    double **uDevice = new double*[NUM_STREAMS];
    double **vDevice = new double*[NUM_STREAMS];
    double **uAverage = new double*[NUM_STREAMS];
    double **vAverage = new double*[NUM_STREAMS];
    double **IxDevice = new double*[NUM_STREAMS];
    double **IyDevice = new double*[NUM_STREAMS];
    double **ItDevice = new double*[NUM_STREAMS];

    // Create streams and allocate memory for each stream
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
        
        // Allocate device memory for each stream
        GPU_ERROR = hipMalloc(&uDevice[i], stream_size);
        GPU_ERROR = hipMalloc(&vDevice[i], stream_size);
        GPU_ERROR = hipMalloc(&uAverage[i], stream_size);
        GPU_ERROR = hipMalloc(&vAverage[i], stream_size);
        GPU_ERROR = hipMalloc(&IxDevice[i], stream_size);
        GPU_ERROR = hipMalloc(&IyDevice[i], stream_size);
        GPU_ERROR = hipMalloc(&ItDevice[i], stream_size);

        // Initialize memory to zero
        hipMemsetAsync(uDevice[i], 0, stream_size, streams[i]);
        hipMemsetAsync(vDevice[i], 0, stream_size, streams[i]);
        hipMemsetAsync(uAverage[i], 0, stream_size, streams[i]);
        hipMemsetAsync(vAverage[i], 0, stream_size, streams[i]);

        // Copy the corresponding portion of data
        int offset = i * rows_per_stream * nx;
        hipMemcpyAsync(IxDevice[i], IxHost.data() + offset, stream_size, 
                       hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(IyDevice[i], IyHost.data() + offset, stream_size, 
                       hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(ItDevice[i], ItHost.data() + offset, stream_size, 
                       hipMemcpyHostToDevice, streams[i]);
    }

    // Compute optical flow
    int currIteration = 0;
    int iterations = 200;
    double alpha = 1;
    while (currIteration < iterations) {
        for (int i = 0; i < NUM_STREAMS; i++) {
            compute_neighbor_average<<<streamGrid, block, 0, streams[i]>>>(
                uDevice[i], vDevice[i], uAverage[i], vAverage[i], 
                nx, rows_per_stream);
        }
        
        // Synchronize all streams after neighbor average computation
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipStreamSynchronize(streams[i]);
        }

        for (int i = 0; i < NUM_STREAMS; i++) {
            horn_schunk<<<streamGrid, block, 0, streams[i]>>>(
                uDevice[i], vDevice[i], uAverage[i], vAverage[i],
                IxDevice[i], IyDevice[i], ItDevice[i], 
                alpha, nx, rows_per_stream);
        }
        
        // Synchronize all streams after horn schunk computation
        for (int i = 0; i < NUM_STREAMS; i++) {
            hipStreamSynchronize(streams[i]);
        }

        currIteration++;
    }

    // Copy results back to host
    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * rows_per_stream * nx;
        hipMemcpyAsync(uHost.data() + offset, uDevice[i], stream_size, 
                       hipMemcpyDeviceToHost, streams[i]);
        hipMemcpyAsync(vHost.data() + offset, vDevice[i], stream_size, 
                       hipMemcpyDeviceToHost, streams[i]);
    }

    // Cleanup
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipFree(uDevice[i]);
        hipFree(vDevice[i]);
        hipFree(uAverage[i]);
        hipFree(vAverage[i]);
        hipFree(IxDevice[i]);
        hipFree(IyDevice[i]);
        hipFree(ItDevice[i]);
        hipStreamDestroy(streams[i]);
    }

    delete[] uDevice;
    delete[] vDevice;
    delete[] uAverage;
    delete[] vAverage;
    delete[] IxDevice;
    delete[] IyDevice;
    delete[] ItDevice;

    // Visualize optical flow
    Mat img_color, flowX, flowY;
    cvtColor(frame1, img_color, COLOR_GRAY2BGR);
    flowX = vectorToMat<double>(uHost, ny, nx, CV_64F);
    flowY = vectorToMat<double>(vHost, ny, nx, CV_64F); 
    drawOpticalFlow(flowX, flowY, img_color);

    Mat flow_vis;
    visualizeFlowHSV(flowX, flowY, flow_vis);

    cout << "Writing optical flow images." << endl;
    imwrite("outputs/CUDA_optical_flow_" + outputname + ".png", img_color);
    imwrite("outputs/CUDA_optical_flow_hsv_" + outputname + ".png", flow_vis);
    return 0;
}